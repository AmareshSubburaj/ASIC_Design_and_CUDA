
#include <stdio.h>
#include "iostream"
#include "fstream"
#include "string"
#include "sstream"
#include "iomanip"
#include <hip/hip_runtime.h>
using namespace std;

//Kernel function to calculate the 2d convolution.

__global__ void
Convolute(const double *Matrix_A, const double *Matrix_B, double *Matrix_C, int row_matrix1, int column_matrix1,int row_matrix2,int column_matrix2,int row_matrix3,int column_matrix3, int numElements)
{
	int Row_C = blockDim.x * blockIdx.x + threadIdx.x;
	int Column_C = blockDim.y * blockIdx.y + threadIdx.y;

    if (Row_C <=row_matrix3 && Column_C <= column_matrix3)
    {
	
	Matrix_C[Row_C*column_matrix3+ Column_C] =0;
		
	for (int Row_B = 0; Row_B<row_matrix2; Row_B++)
	for (int Column_B = 0; Column_B < column_matrix2; Column_B++)
	{

	if ((  (Row_C - Row_B)>=0 && (Row_C - Row_B)< row_matrix1) && ((Column_C - Column_B)>=0 && (Column_C - Column_B) < column_matrix1))
	{		
	
	Matrix_C[Row_C*column_matrix3 + Column_C] += (Matrix_B[Row_B*(column_matrix2) + Column_B] * Matrix_A[(Row_C - Row_B)*(column_matrix1) + (Column_C - Column_B)]);

	}
	
	
    }


  }

}


int main(int argc, char* argv[])
{
    string Filename;

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    //Get the file name from command prompt
    if (argc == 2)
  {

 	 if(argv[1]!=NULL)
   	 Filename = argv[1];

	ifstream iFile;

	iFile.open(Filename.c_str());
	stringstream ss;
	string str_line, str_element;
	bool select_matrix1 = true;
	int column_matrix1 = 0, row_matrix1 = 0, column_matrix2 = 0, row_matrix2 = 0, is_zero = 1;

/***********************get rows and columns*********************************/

	while (getline(iFile, str_line)) {
		ss.str("");
		ss.clear();
		ss << str_line;
		is_zero = str_line.size();

		if ((str_line != "") && (select_matrix1 == true))
		{
			if (column_matrix1 == 0)
				while (ss >> str_element)++column_matrix1;
			++row_matrix1;
		}

		if ((str_line != "") && (select_matrix1 == false))
		{
			if (column_matrix2 == 0)
				while (ss >> str_element)++column_matrix2;
			++row_matrix2;
		}

		if (is_zero == 0)
		{
			select_matrix1 = false;
			is_zero = 1;			
		}
	}

	iFile.close();

/*********************Insert Values into Matrix*********************************/

	//calculate the size of arrays to be allocated in kernel	
	size_t d_A_size = row_matrix1*column_matrix1* sizeof(double);
	size_t d_B_size = row_matrix2*column_matrix2* sizeof(double);
	

	int row_matrix3 = 0, column_matrix3 = 0;
	row_matrix3 = row_matrix1 + row_matrix2 - 1;   
	column_matrix3 = column_matrix1 + column_matrix2 - 1;
	int Matrix_C_size = column_matrix3 * row_matrix3;
	size_t d_C_size = Matrix_C_size*sizeof(double);
	//Total elements.


	double *h_Matrix_C = (double *)malloc(d_C_size);
	double *h_Matrix_A = (double *)malloc(d_A_size); 
	double *h_Matrix_B = (double *)malloc(d_B_size); 

	select_matrix1 = true;
	int Matrix_A_index = 0, Matrix_B_index = 0;

	iFile.open(Filename.c_str());

	while (getline(iFile, str_line)) {

		ss.str("");
		ss.clear();
		ss << str_line;
		is_zero = str_line.size();

		if (is_zero == 0)
		{
			select_matrix1 = false;
			ss.str("");
		}

		{

		if ((str_line != "") && (select_matrix1 == true))
		{

			while (ss >> str_element)
			{
				h_Matrix_A[Matrix_A_index] = atof(str_element.c_str());
				Matrix_A_index++;
			}
		}


		if ((str_line != "") && (select_matrix1 == false))
		{			
			while (ss >> str_element)
			{	

				h_Matrix_B[Matrix_B_index] = atof(str_element.c_str());
				Matrix_B_index++;
			}			
		}
	  }
	}
	iFile.close();

		

/**************************Allocate memory for host arrays*************************/
    double *d_A = NULL;
    err = hipMalloc((void **)&d_A, d_A_size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device Matrix A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    double *d_B = NULL;
    err = hipMalloc((void **)&d_B, d_B_size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device Matrix B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    double *d_C = NULL;
    err = hipMalloc((void **)&d_C, d_C_size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device Matrix C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


/*******************************copy value from host to device***********************/
    err = hipMemcpy(d_A, h_Matrix_A, d_A_size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy Matrix A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_Matrix_B, d_B_size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy Matrix B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch CUDA Kernel
    dim3 threadsPerBlock(16,16);
    dim3 blocksPerGrid((row_matrix3 + threadsPerBlock.x - 1) / threadsPerBlock.x, (column_matrix3 + threadsPerBlock.y - 1) / threadsPerBlock.y);
    Convolute<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, row_matrix1,column_matrix1,row_matrix2,column_matrix2,row_matrix3,column_matrix3,Matrix_C_size);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch Convolute kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_Matrix_C, d_C, d_C_size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Print out the results
	for (int Row_C = 0; Row_C < row_matrix3; Row_C++)
	{
		if(Row_C!=0)		
		cout << "\n";

		for (int Column_C = 0; Column_C < column_matrix3; Column_C++)
		{
			cout <<std::fixed<<std::setprecision(3)<< h_Matrix_C[Row_C*column_matrix3 + Column_C] << " ";
		}
	} 

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device Matrix A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device Matrix B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device Matrix C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory


    free(h_Matrix_A);
    free(h_Matrix_B);
    free(h_Matrix_C);


    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    }
    return 0;
   
}

